/*test.cu*/
#include <stdio.h>
#include "allocations_kernel.cu"
extern void inc_cpu(int* src, int length);
int main (int argc, char **argv){
  int i,j;
  int* a_h;
  int* b_d;
  int* c_h;
  dim3 dG(1);
  dim3 dB(1);

  printf("Checkpoint -2\n");
  i = atoi(argv[1]);
  printf("Checkpoint -1\n");
  a_h = (int*) malloc(i*sizeof(int));
  printf("Checkpoint 0\n");
  hipMalloc((void**)&b_d,i*sizeof(int));
  c_h = (int*) malloc(i*sizeof(int));
  printf("Checkpoint 1\n");
  for(j=0;j<i;j++){
    *(a_h+j) = j;
    printf("The value at a+%d is: %d \n",j,*(a_h+j));
  }
  hipMemcpy(b_d,a_h,i*sizeof(int),hipMemcpyHostToDevice);
  increase<<< 1, 1>>>(b_d);
  hipMemcpy(c_h,b_d,i*sizeof(int),hipMemcpyDeviceToHost);
  for(j=0;j<i;j++)
    printf("The value at c+%d is: %d \n",j,*(c_h+j));
  printf("Checkpoint 3\n"); 
  return 0;
}