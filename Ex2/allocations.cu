/*test.cu*/
#include <stdio.h>
#include "allocations_kernel.cu"
extern void inc_cpu(int* src, int length);
int main (int argc, char **argv){
  int i,j;
  int* a_h;
  int* b_d;
  int* c_h;
  dim3 dG(1);
  dim3 dB(16,8,1);
  i = atoi(argv[1]);
  a_h = (int*) malloc(i*sizeof(int));
  hipMalloc((void**)&b_d,i*sizeof(int));
  c_h = (int*) malloc(i*sizeof(int));
  for(j=0;j<i;j++){
    *(a_h+j) = j;
    printf("The value at a+%d is: %d \n",j,*(a_h+j));
  }
  hipMemcpy(b_d,a_h,i*sizeof(int),hipMemcpyHostToDevice);
  inc_gpu<<< dG, dB>>>(b_d,i);
  hipMemcpy(c_h,b_d,i*sizeof(int),hipMemcpyDeviceToHost);
  for(j=0;j<i;j++)
    printf("The value at c+%d is: %d \n",j,*(c_h+j));
  return 0;
}