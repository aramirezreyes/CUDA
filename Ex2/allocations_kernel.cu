#include "hip/hip_runtime.h"
/* allocations_kernel.cu*/
#include "test_conf.h"
#include <string.h>


__global__ void increase(int* src){
  src[0]++;  
}

__global__ void inc_gpu(int* src, int n){
  int tid = threadIdx.y*blockDim.x + threadIdx.x;
  if(tid<n)
    src[tid]++;
}





