/*matMul.cu*/
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include "matMul_kernel.cu"
extern "C" void matMul_cpu(float *out, float* arrA, float *arrB,int size);
extern "C" bool compareResults(  float *arrA,  float *arrB,  int size,   float eps);
extern "C" void printArray(  float* arr,  int nbelem);

int main(int argc, char **argv){
hipFree(0);
hipEvent_t start,stop;
float msec;
hipEventCreate(&start);
hipEventCreate(&stop);
//get array size from command line argument
if(argc==1){
printf("provide nbelem\n");
exit(0);
}
int nbelem=atoi(argv[1]);

//**** CPU ****
//declare host pointers
float *arrA, *arrB, *arrC;

//allocate on host
arrA = (float*)malloc(nbelem*nbelem*sizeof(float));
arrB = (float*)malloc(nbelem*nbelem*sizeof(float));
arrC = (float*)malloc(nbelem*nbelem*sizeof(float));
for(int i=0; i<nbelem*nbelem;i++)
  arrC[i] = 0;
//initialize on host
srand( time(NULL) );
for(int j=0;j<nbelem;++j){
for(int i=0;i<nbelem;++i){
*(arrA+nbelem*j+i)=rand()%10;
*(arrB+nbelem*j+i)=rand()%10;
}
}

//compute on host, store result in arrC
hipEventRecord(start);
matMul_cpu(arrC,arrA,arrB,nbelem);
hipEventRecord(stop);
hipEventSynchronize(stop);
hipEventElapsedTime(&msec,start,stop);
printf("Time elapsed while computing on cpu: %f\n",msec);
//**** GPU ****
//declare device pointers
float *arrA_d,*arrB_d, *arrC_d;

//allocate on device
gpuErrchk( hipMalloc((void**)&arrA_d,nbelem*nbelem*sizeof(float)) );
gpuErrchk( hipMalloc((void**)&arrB_d,nbelem*nbelem*sizeof(float)) );
gpuErrchk( hipMalloc((void**)&arrC_d,nbelem*nbelem*sizeof(float)) );

//copy input H2D
hipEventRecord(start);
gpuErrchk(  hipMemcpy(arrA_d,arrA,nbelem*nbelem*sizeof(float),hipMemcpyHostToDevice) );
gpuErrchk(  hipMemcpy(arrB_d,arrB,nbelem*nbelem*sizeof(float),hipMemcpyHostToDevice) );
hipEventRecord(stop);
hipEventSynchronize(stop);
hipEventElapsedTime(&msec,start,stop);
printf("Time elapsed while copying data to gpu: %f\n",msec);
//kernel launch parameters
dim3 dG(1);
dim3 dB(BLOCKSIZE,BLOCKSIZE);

//compute on device
hipEventRecord(start);
matMulKernel<<< dG,dB >>>(arrC_d, arrA_d, arrB_d, nbelem);
hipEventRecord(stop);
hipEventSynchronize(stop);
hipEventElapsedTime(&msec,start,stop);
printf("Time elapsed while executing the kernel: %f\n",msec);

//gpuErrchk( hipPeekAtLastError() );
//gpuErrchk( hipDeviceSynchronize() );

//declare and allocate auxiliary host array
float *arrAux;
arrAux = (float*)malloc(nbelem*nbelem*sizeof(float));

//copy result D2H
hipEventRecord(start);
gpuErrchk( hipMemcpy(arrAux,arrC_d,nbelem*nbelem*sizeof(float),hipMemcpyDeviceToHost) );
hipEventRecord(stop);
hipEventSynchronize(stop);
hipEventElapsedTime(&msec,start,stop);
printf("Time elapsed while retrieving data from gpu: %f\n",msec);
//compare hostand device results
if(compareResults(arrC,arrAux,nbelem,1.0e-2))printf("test ok\n");
 else printf("test failed\n");
printf("A\n");
printArray(arrA,nbelem);
printf("B\n");
printArray(arrB,nbelem);
printf("C\n");
printArray(arrC,nbelem);
printf("D\n");
printArray(arrAux,nbelem);
//free memory on host
free(arrA);
free(arrB);
free(arrC);
free(arrAux);

//free memory on device
gpuErrchk( hipFree(arrA_d) );
gpuErrchk( hipFree(arrB_d) );
gpuErrchk( hipFree(arrC_d) );

//reset device
gpuErrchk( hipDeviceReset() );
hipEventDestroy(start);
hipEventDestroy(stop);
}
