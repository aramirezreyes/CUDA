#include "hip/hip_runtime.h"
/*matMul_kernel.cu*/
#include "matMul_conf.h"


//matMul kernel
__global__ void matMulKernel(float *out,  float* arrA,   float* arrB,  int nbelem){
  int tidx =  threadIdx.x;
  int tidy =  threadIdx.y;
  *(out+nbelem*tidx+tidy) = 0;
  if(tidx<nbelem & tidx<nbelem){
	for(int k=0;k<nbelem;k++)
	  *(out+nbelem*tidx+tidy) += *(arrA+nbelem*tidx+k)*(*(arrB+nbelem*k+tidy));
  }
}




